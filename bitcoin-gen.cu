#include <hip/hip_runtime.h>
#include <openssl/evp.h>
#include <openssl/ec.h>
#include <openssl/obj_mac.h>
#include <openssl/bn.h>
#include <openssl/sha.h>
#include <chrono>
#include <iostream>
#include <string>
#include <vector>
#include <iomanip>
#include <sstream>


const int KEY_LENGTH = 32;
const char* BASE58_CHARS = "123456789ABCDEFGHJKLMNPQRSTUVWXYZabcdefghijkmnopqrstuvwxyz";


__global__ void generateRandomNumbers(unsigned char* d_results, int numKeys, int seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < numKeys * KEY_LENGTH) {
        int keyIdx = idx / KEY_LENGTH;
        unsigned int state = seed + idx;
        state ^= (state << 13);
        state ^= (state >> 17);
        state ^= (state << 5);
        state += keyIdx;
        d_results[idx] = static_cast<unsigned char>(state % 256);
    }
}

class BitcoinKeyGenerator {
private:

    static std::string bytesToHex(const unsigned char* data, size_t length) {
        std::stringstream ss;
        ss << std::hex << std::setfill('0');
        for (size_t i = 0; i < length; i++) {
            ss << std::setw(2) << static_cast<int>(data[i]);
        }
        return ss.str();
    }


    static std::string base58Encode(const std::vector<unsigned char>& data) {
        BIGNUM* bn = BN_new();
        BIGNUM* bn_result = BN_new();
        BIGNUM* bn_58 = BN_new();
        std::string result;

        BN_bin2bn(data.data(), data.size(), bn);
        BN_set_word(bn_58, 58);

        while (BN_is_zero(bn) == 0) {
            BN_div(bn, bn_result, bn, bn_58, BN_CTX_new());
            int remainder = BN_get_word(bn_result);
            result = BASE58_CHARS[remainder] + result;
            BN_copy(bn, bn);
        }


        for (size_t i = 0; i < data.size() && data[i] == 0; i++) {
            result = '1' + result;
        }

        BN_free(bn);
        BN_free(bn_result);
        BN_free(bn_58);

        return result;
    }


    static std::string toWIF(const unsigned char* privateKey, size_t keyLength, bool compressed = true) {
        std::vector<unsigned char> wifData;
        

        wifData.push_back(0x80);
        

        wifData.insert(wifData.end(), privateKey, privateKey + keyLength);
        

        if (compressed) {
            wifData.push_back(0x01);
        }
        

        unsigned char hash1[SHA256_DIGEST_LENGTH];
        unsigned char hash2[SHA256_DIGEST_LENGTH];
        
        SHA256_CTX sha256;
        SHA256_Init(&sha256);
        SHA256_Update(&sha256, wifData.data(), wifData.size());
        SHA256_Final(hash1, &sha256);
        
        SHA256_Init(&sha256);
        SHA256_Update(&sha256, hash1, SHA256_DIGEST_LENGTH);
        SHA256_Final(hash2, &sha256);
        

        wifData.insert(wifData.end(), hash2, hash2 + 4);
        

        return base58Encode(wifData);
    }

public:
    static void generateKeysOnCPU(int numKeys) {
        std::cout << "Rozpoczynam generowanie na CPU..." << std::endl;
        auto start = std::chrono::high_resolution_clock::now();
        
        for (int i = 0; i < numKeys; i++) {
            EVP_PKEY_CTX *ctx = nullptr;
            EVP_PKEY *pkey = nullptr;
            BIGNUM *priv_key = nullptr;
            
            try {
                ctx = EVP_PKEY_CTX_new_id(EVP_PKEY_EC, NULL);
                if (!ctx) continue;

                if (EVP_PKEY_keygen_init(ctx) <= 0 ||
                    EVP_PKEY_CTX_set_ec_paramgen_curve_nid(ctx, NID_secp256k1) <= 0 ||
                    EVP_PKEY_keygen(ctx, &pkey) <= 0) {
                    EVP_PKEY_CTX_free(ctx);
                    continue;
                }

                if (EVP_PKEY_get_bn_param(pkey, "priv", &priv_key)) {
                    if (i < 3) { 
                        unsigned char priv_key_bin[32];
                        int bytes = BN_bn2bin(priv_key, priv_key_bin);
                        
                        std::cout << "CPU Klucz " << i + 1 << ":\n";
                        std::cout << "  HEX: " << bytesToHex(priv_key_bin, bytes) << std::endl;
                        std::cout << "  WIF: " << toWIF(priv_key_bin, bytes) << std::endl;
                    }
                    BN_free(priv_key);
                }

                EVP_PKEY_free(pkey);
                EVP_PKEY_CTX_free(ctx);
            } catch (const std::exception& e) {
                std::cerr << "Błąd generowania klucza CPU #" << i << ": " << e.what() << std::endl;
            }

            
            if (i % 1000 == 0) {
                std::cout << "Wygenerowano " << i << " kluczy..." << std::endl;
            }
        }

        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
        double milliseconds = duration.count() / 1000000.0;
        std::cout << "Czas generowania " << numKeys << " kluczy na CPU: " 
                  << milliseconds << " milisekund" << std::endl;
    }

    static void generateKeysOnGPU(int numKeys) {
        std::cout << "Rozpoczynam generowanie na GPU..." << std::endl;
        auto start = std::chrono::high_resolution_clock::now();

        try {
            std::cout << "Alokacja pamięci GPU..." << std::endl;
            unsigned char* d_random;
            size_t totalSize = numKeys * KEY_LENGTH;
            
            if (hipMalloc(&d_random, totalSize) != hipSuccess) {
                throw std::runtime_error("Błąd alokacji pamięci GPU");
            }

            std::cout << "Uruchamianie kernela CUDA..." << std::endl;
            int blockSize = 256;
            int numBlocks = (totalSize + blockSize - 1) / blockSize;
            generateRandomNumbers<<<numBlocks, blockSize>>>(d_random, numKeys, time(NULL));
            
            std::cout << "Kopiowanie wyników z GPU..." << std::endl;
            std::vector<unsigned char> h_random(totalSize);
            if (hipMemcpy(h_random.data(), d_random, totalSize, hipMemcpyDeviceToHost) != hipSuccess) {
                hipFree(d_random);
                throw std::runtime_error("Błąd kopiowania danych z GPU");
            }

            std::cout << "Przetwarzanie wygenerowanych kluczy..." << std::endl;
            for (int i = 0; i < std::min(3, numKeys); i++) {
                BIGNUM *priv_key = BN_bin2bn(&h_random[i * KEY_LENGTH], KEY_LENGTH, NULL);
                if (priv_key) {
                    std::cout << "GPU Klucz " << i + 1 << ":\n";
                    std::cout << "  HEX: " << bytesToHex(&h_random[i * KEY_LENGTH], KEY_LENGTH) << std::endl;
                    std::cout << "  WIF: " << toWIF(&h_random[i * KEY_LENGTH], KEY_LENGTH) << std::endl;
                    BN_free(priv_key);
                }
            }

            hipFree(d_random);

        } catch (const std::exception& e) {
            std::cerr << "Błąd podczas generowania kluczy GPU: " << e.what() << std::endl;
        }

        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
        double milliseconds = duration.count() / 1000000.0;
        std::cout << "Czas generowania " << numKeys << " kluczy na GPU: " 
                  << milliseconds << " milisekund" << std::endl;
    }
};

int main() {
    const int NUM_KEYS = 10000;
    
    std::cout << "Generowanie " << NUM_KEYS << " kluczy Bitcoin - porównanie CPU vs GPU\n" << std::endl;
    
    std::cout << "=== Test CPU ===\n" << std::endl;
    BitcoinKeyGenerator::generateKeysOnCPU(NUM_KEYS);
    
    std::cout << "\n=== Test GPU ===\n" << std::endl;
    BitcoinKeyGenerator::generateKeysOnGPU(NUM_KEYS);
    
    return 0;
}